#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include "myStructs.h"

__device__ const char* conservative[]={"NDEQ","NEQK","STA","MILV","QHRK","NHQK","FYW","HY","MILF"};

__device__ const char* secondConservative[]={"SAG","ATV","CSA","SGND","STPA","STNK","NEQHRK","NDEQHK","SNDEQK","HFY","FVLIM"};

__device__ int checkChar(const char* s, char c)
{
	do{
		if(*s == c) return 1;
	}while (*s++);
	return 0;
}

__device__ int compare(const char* conservative[],const int size,char ch1, char ch2)
{
	for (int i =0; i< size;i++)
    {
		if(checkChar(conservative[i],ch1) && checkChar(conservative[i],ch2))
			return 1;
	}
	return 0;
}

__global__  void createMutant(char *arr,int numElements,int hyphenIdx,int offset,const char *mainSequence,const char *secSequence)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= numElements-1)
		return;
	int g = 0;

	if(i > hyphenIdx){
		g = -1;	
	}
	if(i == hyphenIdx){
		arr[i] = '-';
	}		
	else if(mainSequence[i+offset] == secSequence[i + g])
		arr[i]= '$'; 
	else if(compare(conservative,CONSERVATIVE,mainSequence[i+offset],secSequence[i + g]))
		arr[i]= '%';
	else if(compare(secondConservative,SECOND_CONSERVATIVE,mainSequence[i+offset],secSequence[i + g]))
		arr[i]= '#';
	else
		arr[i] = ' ';
}

char* computeOnGPU(int numElements,int hypenIdx,int offset,const char *mainSequence,const char *secSequence,int tid) 
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

	//Using cuda stream
	const int num_streams = 8;
	hipStream_t stream[num_streams];
	hipStreamCreate(&stream[tid]);

    // Allocate memory on GPU to copy the data from the host
    char *d_Mutant;
	size_t size = numElements * sizeof(char);
    err = hipMalloc((void **)&d_Mutant, size);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	hipMemset(d_Mutant,0,size);
	
	 // Allocate memory on GPU to copy the data from the host
    char *d_mainSequence;
	size_t size_mainSequence = (strlen(mainSequence) + 1) * sizeof(char);
    err = hipMalloc((void **)&d_mainSequence, size_mainSequence);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_mainSequence, mainSequence, size_mainSequence, hipMemcpyHostToDevice);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
	 // Allocate memory on GPU to copy the data from the host
    char *d_secSequence;
	size_t size_secSequence = (strlen(secSequence) + 1) * sizeof(char);
    err = hipMalloc((void **)&d_secSequence, size_secSequence);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_secSequence, secSequence, size_secSequence, hipMemcpyHostToDevice);
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
	createMutant<<<blocksPerGrid, threadsPerBlock, 0, stream[tid]>>>(d_Mutant,numElements,hypenIdx,offset,d_mainSequence,d_secSequence);

    err = hipGetLastError();
    if (err != hipSuccess) 
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	char* result = (char*)malloc(numElements*sizeof(char));
    // Copy the  result from GPU to the host memory.
    err = hipMemcpyAsync(result, d_Mutant, numElements, hipMemcpyDeviceToHost, stream[tid]);

    if (err != hipSuccess) 
    {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU:
    if (hipFree(d_Mutant) != hipSuccess) 
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_mainSequence) != hipSuccess) 
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_secSequence) != hipSuccess) 
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	hipStreamDestroy(stream[tid]);
    return result;
}
